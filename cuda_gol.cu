
#include <hip/hip_runtime.h>
#include <ctime>
#include <stdlib.h>
#include <iostream>
#include <fstream>

__device__ int sum_neighbors(int* board, int r, int c, int n) {
	int sum = 0;
	for(int i = r-1; i < r+2; i++) {
		for(int j = c-1; j < c+2; j++) {
			if((i != r) || (j != c)) {
				sum += board[i*n+j];
			}
		}
	}
	return sum;
}

__global__ void tick(int* board_in, int* board_out, int n) {
	
	//int my_index = (blockDim.x+2)*(blockIdx.x+1) + threadIdx.x+1;
	int row = blockIdx.x+1; //plus one to account for border remaining constant. blocks/threads index the inner matrix
	int col = threadIdx.x+1;
	if(board_in[row*n+col]){
		if(sum_neighbors(board_in, row, col, n)==2 || sum_neighbors(board_in, row, col, n)==3){
			board_out[row*n+col] = 1;
		}
		else{
			board_out[row*n+col] = 0;
		}
	}
	else{
		if(sum_neighbors(board_in, row, col, n)==3){
			board_out[row*n+col] = 1;
		}
		else{
			board_out[row*n+col] = 0;
		}
	}
}
void write_matrix_to_file(int* matrix, int n, int evolution) {
	std::ofstream myfile;
	myfile.open("evolution"+std::to_string(evolution)+".txt");
	for(int i = 0; i < n*n; i++) {
		myfile << matrix[i] << "\n";
	}
}

int main(int argc, char* argv[]) {
	srand((unsigned) time(0));

	int n = atoi(argv[1]);
	int evolutions = atoi(argv[2]);
	int* board_even = new int[n*n];
	int* board_odd = new int[n*n];
	

	//initialize random board
	for(int i = 0; i < n*n; i++) {
		board_even[i] = rand()%2;
		board_odd[i] = board_even[i];//may not be necessary?
	}

	write_matrix_to_file(board_even, n, 0);

	//kill border, border stays dead
	for(int x = 0; x < n; x++) {
		board_even[x] = 0;
		board_even[(n-1)*n + x] = 0;
		board_even[x*n] = 0;
		board_even[x*n+n-1]=0;
		board_odd[x]=0;
		board_odd[(n-1)*n+x]=0;
		board_odd[x*n]=0;
		board_odd[x*n+n-1]=0;
	}

	int *board_even_d, *board_odd_d;
	hipMalloc(&board_even_d, n*n*sizeof(int));
	hipMalloc(&board_odd_d, n*n*sizeof(int));

	hipMemcpy(board_even_d, board_even, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(board_odd_d, board_odd, n*n*sizeof(int), hipMemcpyHostToDevice);
	
	
	//create parameters file
	std::ofstream paramfile;
	paramfile.open("params.txt");
	paramfile << n;
	paramfile << std::endl;
	paramfile << evolutions+1;//total number of state files

	for(int e = 0; e < evolutions; e++) {
		//evolve
		if (e%2==0){
			tick<<<n-2, n-2>>>(board_even_d, board_odd_d, n);
			hipMemcpy(board_odd, board_odd_d,n*n*sizeof(int),hipMemcpyDeviceToHost);
			write_matrix_to_file(board_odd, n, e+1);
		}
		else {
			tick<<<n-2, n-2>>>(board_odd_d, board_even_d, n);
			hipMemcpy(board_even,board_even_d,n*n*sizeof(int),hipMemcpyDeviceToHost);
			write_matrix_to_file(board_even, n, e+1);
		}
	}


	hipFree(board_even_d);
	hipFree(board_odd_d);
	delete[] board_even;
	delete[] board_odd;
	return 0;
}
